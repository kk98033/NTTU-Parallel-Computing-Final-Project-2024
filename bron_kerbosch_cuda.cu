#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <chrono> // For timing

#define MAX_NODES 100

__device__ bool is_clique(const bool *adjacency_matrix, const int *subset, int subset_size, int n)
{
    for (int i = 0; i < subset_size; ++i)
    {
        for (int j = i + 1; j < subset_size; ++j)
        {
            if (!adjacency_matrix[subset[i] * n + subset[j]])
            {
                return false;
            }
        }
    }
    return true;
}

__global__ void find_max_clique_kernel(const bool *adjacency_matrix, int *best_clique, int *max_clique_size, int n)
{
    __shared__ int shared_best_clique[32 * MAX_NODES];
    __shared__ int shared_max_clique_size[32];

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int warp_id = threadIdx.x / 32;

    if (thread_id >= n)
        return;

    int local_max_clique_size = 0;
    int local_best_clique[MAX_NODES];
    int current_clique[MAX_NODES];

    for (int start = thread_id; start < n; start += gridDim.x * blockDim.x)
    {
        int current_clique_size = 0;
        for (int i = start; i < n; ++i)
        {
            current_clique[current_clique_size] = i;
            if (is_clique(adjacency_matrix, current_clique, current_clique_size + 1, n))
            {
                current_clique_size++;
                if (current_clique_size > local_max_clique_size)
                {
                    local_max_clique_size = current_clique_size;
                    for (int j = 0; j < current_clique_size; ++j)
                    {
                        local_best_clique[j] = current_clique[j];
                    }
                }
            }
        }
    }

    if (local_max_clique_size > shared_max_clique_size[warp_id])
    {
        shared_max_clique_size[warp_id] = local_max_clique_size;
        for (int j = 0; j < local_max_clique_size; ++j)
        {
            shared_best_clique[warp_id * MAX_NODES + j] = local_best_clique[j];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0)
    {
        for (int i = 0; i < 32; ++i)
        {
            if (shared_max_clique_size[i] > *max_clique_size)
            {
                *max_clique_size = shared_max_clique_size[i];
                for (int j = 0; j < *max_clique_size; ++j)
                {
                    best_clique[j] = shared_best_clique[i * MAX_NODES + j];
                }
            }
        }
    }
}

void find_max_clique(const std::vector<std::vector<bool>> &adjacency_matrix, std::vector<int> &best_clique, int n)
{
    bool *d_adjacency_matrix;
    int *d_best_clique;
    int *d_max_clique_size;

    size_t matrix_size = n * n * sizeof(bool);
    size_t clique_size = MAX_NODES * sizeof(int);
    hipMalloc(&d_adjacency_matrix, matrix_size);
    hipMalloc(&d_best_clique, clique_size);
    hipMalloc(&d_max_clique_size, sizeof(int));
    hipMemset(d_max_clique_size, 0, sizeof(int));

    bool *h_adjacency_matrix = new bool[n * n];
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            h_adjacency_matrix[i * n + j] = adjacency_matrix[i][j];
        }
    }

    hipMemcpy(d_adjacency_matrix, h_adjacency_matrix, matrix_size, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;

    // 計時開始
    auto start = std::chrono::high_resolution_clock::now();

    find_max_clique_kernel<<<grid_size, block_size>>>(d_adjacency_matrix, d_best_clique, d_max_clique_size, n);
    hipError_t err = hipDeviceSynchronize();

    // 計時結束
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration_ms = end - start;
    std::cout << "Kernel execution time: " << duration_ms.count() << " ms\n";

    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << "\n";
    }

    int h_max_clique_size;
    hipMemcpy(&h_max_clique_size, d_max_clique_size, sizeof(int), hipMemcpyDeviceToHost);

    best_clique.resize(h_max_clique_size);
    hipMemcpy(best_clique.data(), d_best_clique, h_max_clique_size * sizeof(int), hipMemcpyDeviceToHost);

    delete[] h_adjacency_matrix;
    hipFree(d_adjacency_matrix);
    hipFree(d_best_clique);
    hipFree(d_max_clique_size);
}

int main()
{
    // 讀取圖數據
    std::ifstream file("clique_data.txt");
    if (!file)
    {
        std::cerr << "Failed to open file." << std::endl;
        return 1;
    }

    int n;
    file >> n;

    std::vector<std::vector<bool>> adjacency_matrix(n, std::vector<bool>(n, false));
    int u, v;
    while (file >> u >> v)
    {
        adjacency_matrix[u][v] = true;
        adjacency_matrix[v][u] = true;
    }

    std::vector<int> best_clique;
    find_max_clique(adjacency_matrix, best_clique, n);

    std::cout << "Max Clique Size: " << best_clique.size() << "\n";
    std::cout << "Max Clique: ";
    for (int node : best_clique)
    {
        std::cout << node << " ";
    }
    std::cout << "\n";

    return 0;
}
