#include <iostream>
#include <fstream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <chrono>  // For timing


__device__ bool is_clique(const bool* adjacency_matrix, const int* subset, int subset_size, int n) {
    for (int i = 0; i < subset_size; ++i) {
        for (int j = i + 1; j < subset_size; ++j) {
            if (!adjacency_matrix[subset[i] * n + subset[j]]) {
                return false;
            }
        }
    }
    return true;
}

__global__ void find_max_clique_kernel(const bool* adjacency_matrix, int* best_clique, int* max_clique_size, int n) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread_id >= n) return;

    int local_max_clique_size = 0;
    int local_best_clique[100];  // 假設最大節點數為 100

    // printf("Thread %d: Starting kernel execution\n", thread_id);

    for (int start = thread_id; start < n; start += gridDim.x * blockDim.x) {
        int current_clique[100];  // 假設最大節點數為 100
        int current_clique_size = 0;
        for (int i = start; i < n; ++i) {
            current_clique[current_clique_size] = i;
            if (is_clique(adjacency_matrix, current_clique, current_clique_size + 1, n)) {
                current_clique_size++;
                if (current_clique_size > local_max_clique_size) {
                    local_max_clique_size = current_clique_size;
                    for (int j = 0; j < current_clique_size; ++j) {
                        local_best_clique[j] = current_clique[j];
                    }
                }
            }
        }
    }

    // printf("Thread %d: local_max_clique_size = %d\n", thread_id, local_max_clique_size);

    __syncthreads();

    if (local_max_clique_size > atomicMax(max_clique_size, local_max_clique_size)) {
        for (int j = 0; j < local_max_clique_size; ++j) {
            best_clique[j] = local_best_clique[j];
        }
    }

    // printf("Thread %d: Finished kernel execution\n", thread_id);
}

void find_max_clique(const std::vector<std::vector<bool>>& adjacency_matrix, std::vector<int>& best_clique, int n) {
    bool* d_adjacency_matrix;
    int* d_best_clique;
    int* d_max_clique_size;

    size_t matrix_size = n * n * sizeof(bool);
    size_t clique_size = n * sizeof(int);
    hipMalloc(&d_adjacency_matrix, matrix_size);
    hipMalloc(&d_best_clique, clique_size);
    hipMalloc(&d_max_clique_size, sizeof(int));
    hipMemset(d_max_clique_size, 0, sizeof(int));

    bool* h_adjacency_matrix = new bool[n * n];
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            h_adjacency_matrix[i * n + j] = adjacency_matrix[i][j];
        }
    }

    hipMemcpy(d_adjacency_matrix, h_adjacency_matrix, matrix_size, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (n + block_size - 1) / block_size;

    printf("Launching kernel with grid_size=%d, block_size=%d\n", grid_size, block_size);

    // 計時開始
    auto start = std::chrono::high_resolution_clock::now();

    find_max_clique_kernel<<<grid_size, block_size>>>(d_adjacency_matrix, d_best_clique, d_max_clique_size, n);
    hipError_t err = hipDeviceSynchronize();  // Ensure all CUDA operations are finished

    // 計時結束
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration_ms = end - start;
    std::cout << "Kernel execution time: " << duration_ms.count() << " ms\n";

    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }

    int h_max_clique_size;
    hipMemcpy(&h_max_clique_size, d_max_clique_size, sizeof(int), hipMemcpyDeviceToHost);

    // std::cout << "Debug: h_max_clique_size = " << h_max_clique_size << "\n";

    best_clique.resize(h_max_clique_size);
    hipMemcpy(best_clique.data(), d_best_clique, h_max_clique_size * sizeof(int), hipMemcpyDeviceToHost);

    // std::cout << "Debug: CUDA best_clique: ";
    // for (int i = 0; i < h_max_clique_size; ++i) {
    //     std::cout << best_clique[i] << " ";
    // }
    // std::cout << "\n";

    delete[] h_adjacency_matrix;
    hipFree(d_adjacency_matrix);
    hipFree(d_best_clique);
    hipFree(d_max_clique_size);
}

int main() {
    std::ifstream file("clique_data.txt");
    if (!file) {
        std::cerr << "Failed to open file." << std::endl;
        return 1;
    }

    int n;
    file >> n;

    std::vector<std::vector<bool>> adjacency_matrix(n, std::vector<bool>(n, false));
    int u, v;
    while (file >> u >> v) {
        adjacency_matrix[u][v] = true;
        adjacency_matrix[v][u] = true;
    }

    std::vector<int> best_clique;
    find_max_clique(adjacency_matrix, best_clique, n);

    std::cout << "Max Clique Size: " << best_clique.size() << "\n";
    std::cout << "Max Clique: ";
    for (int node : best_clique) {
        std::cout << node << " ";
    }
    std::cout << "\n";

    return 0;
}
